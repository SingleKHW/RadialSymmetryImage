#include "hip/hip_runtime.h"
/*

Project		:	RSImage_GPU
Description	:	GPU implementation of radial symmetry center finding method.
				
				This is a C/C++ translation of the original work,
				<doi:10.1038/nmeth.2071>.

*

Copyright (C) 2016 Janghyun Yoo

This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.

*/
#include "hip/hip_runtime.h"
#include ""
#include "RSImage_GPU.h"

#include <stdlib.h>
#include <cmath>
#include <stdio.h>

#include <Windows.h>
#include <ctime>

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"

#include <stdint.h>
#include <stdio.h>

#define StackSize 32

using namespace std;

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort /* =true */)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void calcDervs(uint8_t *d_image, float *d_du, float *d_dv, size_t *d_x_off, size_t *d_y_off,  size_t *d_width, size_t *d_height, size_t *d_ROIwidth, size_t *d_ROIheight)
{
	// ROI coordinates
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	int j=blockDim.y*blockIdx.y+threadIdx.y;

	if(i>*d_ROIwidth-2 || j>*d_ROIheight-2)  return;

	// Image coordinates
	int abs_i=i+(*d_x_off);
	int abs_j=j+(*d_y_off);


	d_du[(*d_ROIwidth-1)*j+i]=float(d_image[*d_width*(abs_j)+abs_i+1])-float(d_image[*d_width*(abs_j+1)+abs_i]);
	d_dv[(*d_ROIwidth-1)*j+i]=float(d_image[*d_width*(abs_j)+abs_i])-float(d_image[*d_width*(abs_j+1)+abs_i+1]);

}

__global__ void calcDervsF(float *d_du, float *d_duF, float *d_dv, float *d_dvF, size_t *d_ROIwidth, size_t *d_ROIheight, float *d_sw)
{
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	int j=blockDim.y*blockIdx.y+threadIdx.y;


	if(i>*d_ROIwidth-2 || j>*d_ROIheight-2)
	{
		return;
	}

	d_sw[((*d_ROIwidth-1)*j+i)/StackSize]=0;
	d_sw[(*d_ROIwidth)*(*d_ROIheight)*1/StackSize+((*d_ROIwidth-1)*j+i)/StackSize]=0;
	d_sw[(*d_ROIwidth)*(*d_ROIheight)*2/StackSize+((*d_ROIwidth-1)*j+i)/StackSize]=0;
	d_sw[(*d_ROIwidth)*(*d_ROIheight)*3/StackSize+((*d_ROIwidth-1)*j+i)/StackSize]=0;
	d_sw[(*d_ROIwidth)*(*d_ROIheight)*4/StackSize+((*d_ROIwidth-1)*j+i)/StackSize]=0;

	//Smoothing. 3x3 average. Not a boundary pixel
	if (i>0 && i<*d_ROIwidth-2 && j>0 && j<*d_ROIheight-2)
	{
			d_duF[(*d_ROIwidth-1)*j+i]=(	d_du[(*d_ROIwidth-1)*(j-1)+i-1]+	d_du[(*d_ROIwidth-1)*(j-1)+i]+	d_du[(*d_ROIwidth-1)*(j-1)+i+1]+ \
													d_du[(*d_ROIwidth-1)*j+i-1]+	d_du[(*d_ROIwidth-1)*j+i]+		d_du[(*d_ROIwidth-1)*j+i+1]+ \
													d_du[(*d_ROIwidth-1)*(j+1)+i-1]+	d_du[(*d_ROIwidth-1)*(j+1)+i]+	d_du[(*d_ROIwidth-1)*(j+1)+i+1])/9;
			d_dvF[(*d_ROIwidth-1)*j+i]=(	d_dv[(*d_ROIwidth-1)*(j-1)+i-1]+	d_dv[(*d_ROIwidth-1)*(j-1)+i]+	d_dv[(*d_ROIwidth-1)*(j-1)+i+1]+ \
													d_dv[(*d_ROIwidth-1)*j+i-1]+	d_dv[(*d_ROIwidth-1)*j+i]+		d_dv[(*d_ROIwidth-1)*j+i+1]+ \
													d_dv[(*d_ROIwidth-1)*(j+1)+i-1]+	d_dv[(*d_ROIwidth-1)*(j+1)+i]+	d_dv[(*d_ROIwidth-1)*(j+1)+i+1])/9;
			return;
	}
	
	//option2: no smoothing edges
	/*
	d_duF[(*d_ROIwidth-1)*j+i]=d_du[(*d_ROIwidth-1)*j+i];
	d_dvF[(*d_ROIwidth-1)*j+i]=d_dv[(*d_ROIwidth-1)*j+i];

	return;
	*/

	//Smoothing edges
	if (j==0)
	{
		if(i==0) //left upper corner
		{
			d_duF[0]=(d_du[0]+	d_du[1]+	d_du[(*d_ROIwidth-1)*1+0]+	d_du[(*d_ROIwidth-1)*1+1])/4;
			d_dvF[0]=(d_dv[0]+	d_dv[1]+	d_dv[(*d_ROIwidth-1)*1+0]+	d_dv[(*d_ROIwidth-1)*1+1])/4;
			return;
		}

		if(i==*d_ROIwidth-2) // right upper corner
		{
			d_duF[i]=(d_du[i-1]+	d_du[i]+	d_du[(*d_ROIwidth-1)*1+i-1]+	d_du[(*d_ROIwidth-1)*1+i])/4;
			d_dvF[i]=(d_dv[i-1]+	d_dv[i]+	d_dv[(*d_ROIwidth-1)*1+i-1]+	d_dv[(*d_ROIwidth-1)*1+i])/4;
			return;
		}

		//the top line
		if(i<*d_ROIwidth-1)
		{
			d_duF[(*d_ROIwidth-1)*j+i]=(	d_du[i-1]+	d_du[i]+		d_du[i+1]+	\
													d_du[(*d_ROIwidth-1)*1+i-1]+	d_du[(*d_ROIwidth-1)*1+i]+	d_du[(*d_ROIwidth-1)*1+i+1])/6;
			d_dvF[(*d_ROIwidth-1)*j+i]=(	d_dv[i-1]+	d_dv[i]+		d_dv[i+1]+	\
													d_dv[(*d_ROIwidth-1)*1+i-1]+	d_dv[(*d_ROIwidth-1)*1+i]+	d_dv[(*d_ROIwidth-1)*1+i+1])/6;
			return;
		}
	}

	if (j==*d_ROIwidth-2)
	{
		if(i==0) //bottom left
		{
			d_duF[(*d_ROIwidth-1)*(j)+(0)]=(	d_du[(*d_ROIwidth-1)*(j-1)+(0)]+	d_du[(*d_ROIwidth-1)*(j-1)+(1)]+	\
													d_du[(*d_ROIwidth-1)*(j)+(0)]+		d_du[(*d_ROIwidth-1)*(j)+(1)])/4;
			d_dvF[(*d_ROIwidth-1)*(j)+(0)]=(	d_dv[(*d_ROIwidth-1)*(j-1)+(0)]+	d_dv[(*d_ROIwidth-1)*(j-1)+(1)]+	\
													d_dv[(*d_ROIwidth-1)*(j)+(0)]+		d_dv[(*d_ROIwidth-1)*(j)+(1)])/4;
			return;
		}

		if(i==*d_ROIwidth-2) //bottom right
		{
			d_duF[(*d_ROIwidth-1)*j+i]=(	d_du[(*d_ROIwidth-1)*(j-1)+i-1]+	d_du[(*d_ROIwidth-1)*(j-1)+i]+		\
													d_du[(*d_ROIwidth-1)*j+i-1]+	d_du[(*d_ROIwidth-1)*j+i])/4;
			d_dvF[(*d_ROIwidth-1)*j+i]=(	d_dv[(*d_ROIwidth-1)*(j-1)+i-1]+	d_dv[(*d_ROIwidth-1)*(j-1)+i]+	\
													d_dv[(*d_ROIwidth-1)*(j-1)+i-1]+	d_dv[(*d_ROIwidth-1)*j+i])/4;
			return;
		}

		if(i<*d_ROIwidth-1)// the bottom line
		{
			d_duF[(*d_ROIwidth-1)*j+i]=(	d_du[(*d_ROIwidth-1)*(j-1)+i-1]+		d_du[(*d_ROIwidth-1)*(j-1)+i]+	d_du[(*d_ROIwidth-1)*(j-1)+i+1]+	\
												d_du[(*d_ROIwidth-1)*j+i-1]+		d_du[(*d_ROIwidth-1)*j+i]+		d_du[(*d_ROIwidth-1)*j+i+1])/6;
			d_dvF[(*d_ROIwidth-1)*j+i]=(	d_dv[(*d_ROIwidth-1)*(j-1)+i-1]+		d_dv[(*d_ROIwidth-1)*(j-1)+i]+	d_dv[(*d_ROIwidth-1)*(j-1)+i+1]+	\
												d_dv[(*d_ROIwidth-1)*j+i-1]+		d_dv[(*d_ROIwidth-1)*j+i]+		d_dv[(*d_ROIwidth-1)*j+i+1])/6;
			return;
		}
	}

	if(i==0) //the left line
	{
		d_duF[(*d_ROIwidth-1)*(j)+(0)]=(	d_du[(*d_ROIwidth-1)*(j-1)+(0)]+	d_du[(*d_ROIwidth-1)*(j-1)+(1)]+ \
												d_du[(*d_ROIwidth-1)*(j)+(0)]+		d_du[(*d_ROIwidth-1)*(j)+(1)]+ \
												d_du[(*d_ROIwidth-1)*(j+1)+(0)]+	d_du[(*d_ROIwidth-1)*(j+1)+(1)])/6;
		d_dvF[(*d_ROIwidth-1)*(j)+(0)]=(	d_dv[(*d_ROIwidth-1)*(j-1)+(0)]+	d_dv[(*d_ROIwidth-1)*(j-1)+(1)]+ \
												d_dv[(*d_ROIwidth-1)*(j)+(0)]+		d_dv[(*d_ROIwidth-1)*(j)+(1)]+ \
												d_dv[(*d_ROIwidth-1)*(j+1)+(0)]+	d_dv[(*d_ROIwidth-1)*(j+1)+(1)])/6;
		return;
	}

	if(i<*d_ROIwidth-1 && j<*d_ROIheight-1)//the right line
	{
		d_duF[(*d_ROIwidth-1)*j+i]=(	d_du[(*d_ROIwidth-1)*(j-1)+i-1]+		d_du[(*d_ROIwidth-1)*(j-1)+i]+	\
												d_du[(*d_ROIwidth-1)*j+i-1]+		d_du[(*d_ROIwidth-1)*j+i]+		\
												d_du[(*d_ROIwidth-1)*(j+1)+i-1]+		d_du[(*d_ROIwidth-1)*(j+1)+i])/6;
		d_dvF[(*d_ROIwidth-1)*j+i]=(	d_dv[(*d_ROIwidth-1)*(j-1)+i-1]+		d_dv[(*d_ROIwidth-1)*(j-1)+i]+	\
												d_dv[(*d_ROIwidth-1)*j+i-1]+		d_dv[(*d_ROIwidth-1)*j+i]+		\
											d_dv[(*d_ROIwidth-1)*(j+1)+i-1]+		d_dv[(*d_ROIwidth-1)*(j+1)+i])/6;
		return;
	}
}

__global__ void calcGrads(float *d_duF, float *d_dvF, size_t *d_ROIwidth, size_t *d_ROIheight, float *d_x_c_old, float *d_y_c_old, float *sw)
{
	int i=blockDim.x*blockIdx.x+threadIdx.x;
	int j=blockDim.y*blockIdx.y+threadIdx.y;

	if(i>(*d_ROIwidth-2) || j>(*d_ROIheight-2)) return;

	float gradDenominator=d_duF[(*d_ROIwidth-1)*j+i]-d_dvF[(*d_ROIwidth-1)*j+i];
	float m=1, b=0;

	if (gradDenominator==0.f)
		m=-(gradDenominator+2*d_dvF[(*d_ROIwidth-1)*j+i])/minF;
	else
		m=-(gradDenominator+2*d_dvF[(*d_ROIwidth-1)*j+i])/gradDenominator;

	float gridY=-float(*d_ROIheight-1)/2.0f+0.5f+float(j);
	float gridX=-float(*d_ROIwidth-1)/2.0f+0.5f+float(i);
	float gradMag=d_duF[(*d_ROIwidth-1)*j+i]*d_duF[(*d_ROIwidth-1)*j+i]+d_dvF[(*d_ROIwidth-1)*j+i]*d_dvF[(*d_ROIwidth-1)*j+i];

	b=gridY-m*gridX;

	float wm2p1=gradMag/sqrt((gridX-*d_x_c_old)*(gridX-*d_x_c_old)+(gridY-*d_y_c_old)*(gridY-*d_y_c_old))/(m*m+1);

	/*
	sw[(*d_ROIwidth-1)*j+i]=wm2p1;
	smmw[(*d_ROIwidth-1)*j+i]=m*m*wm2p1;
	smw[(*d_ROIwidth-1)*j+i]=m*wm2p1;
	smbw[(*d_ROIwidth-1)*j+i]=m*b*wm2p1;
	sbw[(*d_ROIwidth-1)*j+i]=b*wm2p1;
	*/

	atomicAdd(&sw[((*d_ROIwidth-1)*j+i)/StackSize],wm2p1); //sw
	atomicAdd(&sw[(*d_ROIwidth)*(*d_ROIheight)/StackSize+((*d_ROIwidth-1)*j+i)/StackSize],m*m*wm2p1); //smmw
	atomicAdd(&sw[(*d_ROIwidth)*(*d_ROIheight)*2/StackSize+((*d_ROIwidth-1)*j+i)/StackSize],m*wm2p1); //smw
	atomicAdd(&sw[(*d_ROIwidth)*(*d_ROIheight)*3/StackSize+((*d_ROIwidth-1)*j+i)/StackSize],m*b*wm2p1); //smbw
	atomicAdd(&sw[(*d_ROIwidth)*(*d_ROIheight)*4/StackSize+((*d_ROIwidth-1)*j+i)/StackSize],b*wm2p1); //sbw
}

void calcCenter(RSImage_GPU * RSImage, hipStream_t stream)
{
	//init variables
	RSImage->sw=0;
	RSImage->smmw=0;
	RSImage->smw=0;
	RSImage->smbw=0;
	RSImage->sbw=0;

	//order = {sw, smmw, smw, smbw, sbw}
	for(size_t i=0;i<((RSImage->ROIwidth-1)*(RSImage->ROIheight-1)-1)/StackSize+1;i++)
	{
		RSImage->sw+=RSImage->h_sw[i];
	}
	for(size_t i=0;i<((RSImage->ROIwidth-1)*(RSImage->ROIheight-1)-1)/StackSize+1;i++)
	{
		RSImage->smmw+=RSImage->h_sw[(RSImage->ROIwidth)*(RSImage->ROIheight)/StackSize+i];
	}
	for(size_t i=0;i<((RSImage->ROIwidth-1)*(RSImage->ROIheight-1)-1)/StackSize+1;i++)
	{
		RSImage->smw+=RSImage->h_sw[(RSImage->ROIwidth)*(RSImage->ROIheight)/StackSize*2+i];
	}
	for(size_t i=0;i<((RSImage->ROIwidth-1)*(RSImage->ROIheight-1)-1)/StackSize+1;i++)
	{
		RSImage->smbw+=RSImage->h_sw[(RSImage->ROIwidth)*(RSImage->ROIheight)/StackSize*3+i];
	}
	for(size_t i=0;i<((RSImage->ROIwidth-1)*(RSImage->ROIheight-1)-1)/StackSize+1;i++)
	{
		RSImage->sbw+=RSImage->h_sw[(RSImage->ROIwidth)*(RSImage->ROIheight)/StackSize*4+i];
	}

	RSImage->det=RSImage->smw*RSImage->smw-RSImage->smmw*RSImage->sw;
	RSImage->h_x_c=(RSImage->smbw*RSImage->sw-RSImage->smw*RSImage->sbw)/RSImage->det;
	RSImage->h_y_c=(RSImage->smbw*RSImage->smw-RSImage->smmw*RSImage->sbw)/RSImage->det;

	//update the previous center
	// should relay the position to the next RSImage

	//cudaCheck(hipMemcpyAsync(RSImage->d_x_c_old,&RSImage->h_x_c,sizeof*RSImage->d_x_c_old,hipMemcpyHostToDevice,stream));
	//cudaCheck(hipMemcpyAsync(RSImage->d_y_c_old,&RSImage->h_y_c,sizeof*RSImage->d_y_c_old,hipMemcpyHostToDevice,stream));
}


void updateRSImageCenter(RSImage_GPU * RSImage, dim3 grid, dim3 block,hipStream_t stream){
	calcDervs<<<grid,block,0,stream>>>(RSImage->d_image,RSImage->d_du,RSImage->d_dv,RSImage->d_x_off,RSImage->d_y_off,RSImage->d_width, RSImage->d_height, RSImage->d_ROIwidth, RSImage->d_ROIheight);
	calcDervsF<<<grid,block,0,stream>>>(RSImage->d_du,RSImage->d_duF,RSImage->d_dv,RSImage->d_dvF, RSImage->d_ROIwidth, RSImage->d_ROIheight, RSImage->d_sw);
	calcGrads<<<grid,block,0,stream>>>(RSImage->d_duF, RSImage->d_dvF, RSImage->d_ROIwidth, RSImage->d_ROIheight, RSImage->d_x_c_old, RSImage->d_y_c_old, RSImage->d_sw);
}

//transfer image from host to device
void transferRSImageHtoD(RSImage_GPU * RSImage, hipStream_t stream)
{
	cudaCheck(hipMemcpyAsync(RSImage->d_image,RSImage->h_image,RSImage->width*RSImage->height*sizeof*RSImage->d_image,hipMemcpyHostToDevice,stream));
} 

//transfer derivative images from device to host
void transferRSImageDtoH(RSImage_GPU * RSImage, hipStream_t stream)
{
	cudaCheck(hipMemcpyAsync(RSImage->h_sw,RSImage->d_sw,(RSImage->ROIwidth)*(RSImage->ROIheight)*sizeof*RSImage->d_sw/StackSize*5,hipMemcpyDeviceToHost,stream));
	//cudaCheck(hipMemcpyAsync(RSImage->h_smmw,RSImage->d_smmw,(RSImage->ROIwidth-1)*(RSImage->ROIheight-1)*sizeof*RSImage->d_smmw,hipMemcpyDeviceToHost,stream));
	//cudaCheck(hipMemcpyAsync(RSImage->h_smw,RSImage->d_smw,(RSImage->ROIwidth-1)*(RSImage->ROIheight-1)*sizeof*RSImage->h_smw,hipMemcpyDeviceToHost,stream));
	//cudaCheck(hipMemcpyAsync(RSImage->h_smbw,RSImage->d_smbw,(RSImage->ROIwidth-1)*(RSImage->ROIheight-1)*sizeof*RSImage->d_smbw,hipMemcpyDeviceToHost,stream));
	//cudaCheck(hipMemcpyAsync(RSImage->h_sbw,RSImage->d_sbw,(RSImage->ROIwidth-1)*(RSImage->ROIheight-1)*sizeof*RSImage->d_sbw,hipMemcpyDeviceToHost,stream));
}

void initRSImage(RSImage_GPU *RSImage, uint8_t *h_image, size_t width,size_t height, size_t ROIwidth, size_t ROIheight, size_t x_off, size_t y_off, float x_c_old/* =0.1f */, float y_c_old /* =0.1f */)
{
	//bind image
	RSImage->h_image=h_image;
	RSImage->ROIwidth=ROIwidth;
	RSImage->ROIheight=ROIheight;
	RSImage->width=width;
	RSImage->height=height;

	//the image info
	cudaCheck(hipMalloc(&RSImage->d_width,sizeof*RSImage->d_width));
	cudaCheck(hipMalloc(&RSImage->d_height,sizeof*RSImage->d_height));
	cudaCheck(hipMalloc(&RSImage->d_ROIwidth,sizeof*RSImage->d_ROIwidth));
	cudaCheck(hipMalloc(&RSImage->d_ROIheight,sizeof*RSImage->d_ROIheight));

	cudaCheck(hipMemcpy(RSImage->d_width,&width,sizeof(width),hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(RSImage->d_height,&height,sizeof(height),hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(RSImage->d_ROIwidth,&ROIwidth,sizeof(ROIwidth),hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(RSImage->d_ROIheight,&ROIheight,sizeof(ROIheight),hipMemcpyHostToDevice));

	//bead position
	cudaCheck(hipMalloc((void**)&RSImage->d_x_off,sizeof*RSImage->d_x_off));
	cudaCheck(hipMalloc((void**)&RSImage->d_y_off,sizeof*RSImage->d_y_off));

	cudaCheck(hipMemcpy(RSImage->d_x_off,&x_off,sizeof*RSImage->d_x_off,hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(RSImage->d_y_off,&y_off,sizeof*RSImage->d_y_off,hipMemcpyHostToDevice));

	//device images
	cudaCheck(hipMalloc((void**)&RSImage->d_image,width*height*sizeof*RSImage->d_image));
	cudaCheck(hipMalloc((void**)&RSImage->d_du,(ROIwidth-1)*(ROIheight-1)*sizeof*RSImage->d_du));
	cudaCheck(hipMalloc((void**)&RSImage->d_dv,(ROIwidth-1)*(ROIheight-1)*sizeof*RSImage->d_dv));
	cudaCheck(hipMalloc((void**)&RSImage->d_duF,(ROIwidth-1)*(ROIheight-1)*sizeof*RSImage->d_duF));
	cudaCheck(hipMalloc((void**)&RSImage->d_dvF,(ROIwidth-1)*(ROIheight-1)*sizeof*RSImage->d_dvF));
	cudaCheck(hipMalloc((void**)&RSImage->d_grads,(ROIwidth-1)*(ROIheight-1)*sizeof*RSImage->d_grads));

	//host images for center calculation
	cudaCheck(hipHostMalloc(&RSImage->h_sw,(RSImage->ROIwidth)*(RSImage->ROIheight)*sizeof*RSImage->h_sw/StackSize*5));//merge 5 variables into an array
//	cudaCheck(hipHostMalloc(&RSImage->h_smw,(RSImage->ROIwidth-1)*(RSImage->ROIheight-1)*sizeof*RSImage->h_smw));
//	cudaCheck(hipHostMalloc(&RSImage->h_smmw,(RSImage->ROIwidth-1)*(RSImage->ROIheight-1)*sizeof*RSImage->h_smmw));
//	cudaCheck(hipHostMalloc(&RSImage->h_smbw,(RSImage->ROIwidth-1)*(RSImage->ROIheight-1)*sizeof*RSImage->h_smbw));
//	cudaCheck(hipHostMalloc(&RSImage->h_sbw,(RSImage->ROIwidth-1)*(RSImage->ROIheight-1)*sizeof*RSImage->h_sbw));

	//the previous center
	cudaCheck(hipMalloc((void**)&RSImage->d_x_c_old,sizeof*RSImage->d_x_c_old))
	cudaCheck(hipMalloc((void**)&RSImage->d_y_c_old,sizeof*RSImage->d_y_c_old));

	cudaCheck(hipMemcpy(RSImage->d_x_c_old,&x_c_old,sizeof*RSImage->d_x_c_old,hipMemcpyHostToDevice));
	cudaCheck(hipMemcpy(RSImage->d_y_c_old,&y_c_old,sizeof*RSImage->d_y_c_old,hipMemcpyHostToDevice));

	cudaCheck(hipMalloc(&RSImage->d_sw,(RSImage->ROIwidth)*(RSImage->ROIheight)*sizeof*RSImage->d_sw/StackSize*5)); //merge 5 variables into an array
//	cudaCheck(hipMalloc(&RSImage->d_smw,(RSImage->ROIwidth-1)*(RSImage->ROIheight-1)*sizeof*RSImage->d_smw));
//	cudaCheck(hipMalloc(&RSImage->d_smmw,(RSImage->ROIwidth-1)*(RSImage->ROIheight-1)*sizeof*RSImage->d_smmw));
//	cudaCheck(hipMalloc(&RSImage->d_smbw,(RSImage->ROIwidth-1)*(RSImage->ROIheight-1)*sizeof*RSImage->d_smbw));
//	cudaCheck(hipMalloc(&RSImage->d_sbw,(RSImage->ROIwidth-1)*(RSImage->ROIheight-1)*sizeof*RSImage->d_sbw));
}

void freeRSImage(RSImage_GPU *RSImage)
{
	//the image info
	cudaCheck(hipFree(RSImage->d_width));
	cudaCheck(hipFree(RSImage->d_height));
	cudaCheck(hipFree(RSImage->d_ROIwidth));
	cudaCheck(hipFree(RSImage->d_ROIheight));
	//bead position
	cudaCheck(hipFree(RSImage->d_x_off));
	cudaCheck(hipFree(RSImage->d_y_off));

	//device images
	cudaCheck(hipFree(RSImage->d_image));

	cudaCheck(hipFree(RSImage->d_du));
	cudaCheck(hipFree(RSImage->d_dv));
	cudaCheck(hipFree(RSImage->d_duF));
	cudaCheck(hipFree(RSImage->d_dvF));
	cudaCheck(hipFree(RSImage->d_grads));

	//host derivative images
	cudaCheck(hipHostFree(RSImage->h_sw));
/*	cudaCheck(hipHostFree(RSImage->h_smw));
	cudaCheck(hipHostFree(RSImage->h_smmw));
	cudaCheck(hipHostFree(RSImage->h_smbw));
	cudaCheck(hipHostFree(RSImage->h_sbw));	*/

	//the previous center
	cudaCheck(hipFree(RSImage->d_x_c_old));
	cudaCheck(hipFree(RSImage->d_y_c_old));

	cudaCheck(hipFree(RSImage->d_sw));
/*	cudaCheck(hipFree(RSImage->d_smmw));
	cudaCheck(hipFree(RSImage->d_smw));
	cudaCheck(hipFree(RSImage->d_smbw));
	cudaCheck(hipFree(RSImage->d_sbw)); */
}